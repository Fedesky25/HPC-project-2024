#include "hip/hip_runtime.h"
//
// Created by Sofyh02 on 10/08/2024.
//

#include "particle_generator.cuh"
#include <cuda/std/cmath>
#include <chrono>
#include <hiprand.h>
#include <random>
#include <thrust/sort.h>
#include <omp.h>
#include "lower_bound.cuh"

#include <iostream>
#include <iomanip>

#define LLOYD_ITERATIONS 12

#define SETUP_CPU \
    auto sites = (complex_t*) malloc(N * sizeof(complex_t));           \
    auto n_density = 128*(int64_t)N;                                   \
    auto density = (complex_t*) malloc(n_density * sizeof(complex_t)); \
    auto nearest = (uint32_t*) malloc(n_density * sizeof(uint32_t));   \
    auto count = (uint64_t*) malloc(N * sizeof(uint64_t));

#define PRINT_INITIAL std::cout << "Random initial numbers (" << N << " sites, " << n_density << " density points)";

/**
 * Generates M random complex numbers in the rectangle
 * @param z1 lower-left vertex
 * @param z2 upper-right vertex
 * @param rdm array of points
 * @param M number of sites
 */
void rand_complex(complex_t z1, complex_t z2, complex_t * rdm, uint64_t M) {
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_real_distribution<double> distribution(0.0,1.0);
    for(uint64_t i=0; i<M; i++){
        rdm[i].real(real(z1) + distribution(generator)*(real(z2)-real(z1)));
        rdm[i].imag(imag(z1) + distribution(generator)*(imag(z2)-imag(z1)));
    }
}


complex_t* particles_serial(complex_t z1, complex_t z2, uint32_t N){
    SETUP_CPU PRINT_INITIAL timers(3) tick(0)
    rand_complex(z1, z2, sites, N);
    rand_complex(z1, z2, density, n_density);
    tock_ms(0) std::cout << " generated in " << t_elapsed << "ms" << std::endl;

    float times[2];
    std::cout << "Lloyd's algorithm:  i | t  (s) | n. c. | s. u." << std::endl << std::fixed;
    tick(0)
    for(uint16_t i=0; i<LLOYD_ITERATIONS; i++){  // Iterating to convergence
        tick(1) tick(2)
        for(uint64_t j=0; j<n_density; j++){ // Iterating on density points
            double current, min = INFINITY;
            for(uint64_t k=0; k<N; k++){  // Iterating on sites to save the nearest site to each density point
                current = cuda::std::norm(density[j]-sites[k]);
                if(current < min){
                   nearest[j] = k;
                   min = current;
               }
            }
        } // Here nearest[] has been filled in
        tock_s(2) times[0] = t_elapsed; tick(2)
        for(int64_t k=0; k<N; k++) {
            sites[k] = 0;
            count[k] = 0;
        }
        for(int64_t j=0; j<n_density; j++){ // Iterating on nearest
            sites[nearest[j]] += density[j];
            count[nearest[j]] ++;
        }
        for (int64_t k = 0; k < N; k++) {
            if (count[k] == 0) rand_complex(z1, z2, sites + k, 1);
            else sites[k] /= (double) count[k];
        }
        tock_s(2) times[1] = t_elapsed; tock_s(1)
        float m = 100.0f / t_elapsed;
        std::cout << "                   " << std::setw(2) << i+1
                  << " | " << std::setw(6) << std::setprecision(3) << t_elapsed
                  << " | " << std::setw(5) << std::setprecision(2) << times[0]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[1]*m << std::endl;
//        for(uint64_t k=0; k<N; k++){ // Iterating on sites
//            double ctr = 0;
//            sites[k] = 0;
//            for(uint64_t j=0; j<n_density; j++){ // Iterating on nearest
//                if(nearest[j] == k){  // Finding density points associated to the k-th site
//                    sites[k] += density[j];
//                    ctr++;
//                }
//            }
//            if(ctr != 0) sites[k] /= ctr;
//        }
    }
    tock_s(0) std::cout << "  :: total " << std::setprecision(3) << t_elapsed << 's' << std::endl;
    free(density);
    free(nearest);
    free(count);
    return sites;
}


void rand_complex_omp(
        complex_t min, complex_t max,
        complex_t * sites, uint32_t N_sites,
        complex_t * density, uint64_t N_density
) {
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    #pragma omp parallel
    {
        std::default_random_engine generator(seed + omp_get_thread_num());
        std::uniform_real_distribution<double> dist_real(min.real(), max.real());
        std::uniform_real_distribution<double> dist_imag(min.imag(), max.imag());
        #pragma omp for schedule(static)
        for(int32_t i=0; i<N_sites; i++){
            sites[i].real(dist_real(generator));
            sites[i].imag(dist_imag(generator));
        }
        #pragma omp for schedule(static)
        for(int64_t i=0; i<N_density; i++){
            density[i].real(dist_real(generator));
            density[i].imag(dist_imag(generator));
        }
    };
}


complex_t* particles_omp(complex_t z1, complex_t z2, uint32_t N){
    SETUP_CPU PRINT_INITIAL timers(3) tick(0)
    rand_complex_omp(z1, z2, sites, N, density, n_density);
    tock_ms(0) std::cout << " generated in " << t_elapsed << "ms" << std::endl;

    float times[2];
    std::cout << "Lloyd's algorithm:  i | t  (s) | n. c. | s. u.    using " << std::endl << std::fixed;
    tick(0)
    for(int16_t i=0; i<LLOYD_ITERATIONS; i++){  // Iterating to convergence
        tick(1) tick(2)
        #pragma omp parallel for shared(nearest, density, sites) schedule(static)
        for (int64_t j = 0; j < n_density; j++) { // Iterating on density points
            double current, min = INFINITY;
            for (int64_t k = 0; k < N; k++) {  // Iterating on sites to save the nearest site to each density point
                current = cuda::std::norm(density[j] - sites[k]);
                if (current < min) {
                    nearest[j] = k;
                    min = current;
                }
            }
        }
        tock_s(2) times[0] = t_elapsed; tick(2)
        for(int64_t k=0; k<N; k++) {
            sites[k] = 0;
            count[k] = 0;
        }
        for(int64_t j=0; j<n_density; j++){ // Iterating on nearest
            sites[nearest[j]] += density[j];
            count[nearest[j]] ++;
        }
        #pragma omp parallel for shared(sites, count) schedule(static)
        for (int64_t k = 0; k < N; k++) {
            if (count[k] == 0) rand_complex(z1, z2, sites + k, 1);
            else sites[k] /= (double) count[k];
        }
        tock_s(2) times[1] = t_elapsed; tock_s(1)
        float m = 100.0f / t_elapsed;
        std::cout << "                   " << std::setw(2) << i+1
                  << " | " << std::setw(6) << std::setprecision(3) << t_elapsed
                  << " | " << std::setw(5) << std::setprecision(2) << times[0]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[1]*m << std::endl;
    }
    free(density);
    free(nearest);
    free(count);
    tock_s(0)
    std::cout << "  :: total " << std::setprecision(3) << t_elapsed << 's' << std::endl;
    return sites;
}

__global__ void compute_nearest(
        complex_t * density_points, int64_t N_density,
        complex_t * sites, uint32_t N_sites,
        uint32_t * nearest
){
    auto index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index >= N_density) return;
    double current, min = INFINITY;
    complex_t z = density_points[index];
    uint32_t n;
    for (uint32_t k = 0; k < N_sites; k++) {  // Iterating on sites to save the nearest site to each density point
        current = cuda::std::norm(z - sites[k]);
        if (current < min) {
            n = k;
            min = current;
        }
    }
    nearest[index] = n;
}

complex_t* particles_mixed(complex_t z1, complex_t z2, uint32_t N){
    SETUP_CPU
    complex_t *d_density, *d_sites;
    uint32_t *d_nearest;
    hipMalloc((void **)&d_density, n_density * sizeof (complex_t));
    hipMalloc((void **)&d_sites, N * sizeof (complex_t));
    hipMalloc((void **)&d_nearest, n_density * sizeof (uint32_t));

    PRINT_INITIAL timers(3) tick(0)
    rand_complex_omp(z1, z2, sites, N, density, n_density);
    tock_ms(0) std::cout << " generated in " << t_elapsed << "ms" << std::endl;

    float times[4];
    auto M = ((n_density-1) >> 10) + 1; // (n_density + 1023) / 1024 = (n_density-1)/ 2^(10)
    hipMemcpy(d_density, density, n_density * sizeof (complex_t), hipMemcpyHostToDevice);

    std::cout << "Lloyd's algorithm:  i | t (ms) | s. -> | n. c. | n. <- | s. u." << std::endl << std::fixed;
    tick(0)

    for(int16_t i=0; i<LLOYD_ITERATIONS; i++){  // Iterating to convergence
        tick(1) tick(2)
        hipMemcpy(d_sites, sites, N * sizeof (complex_t), hipMemcpyHostToDevice);
        tock_ms(2) times[0] = t_elapsed; tick(2)
        compute_nearest<<<M, 1024>>>(d_density, n_density, d_sites, N, d_nearest);
        tock_ms(2) times[1] = t_elapsed; tick(2)
        hipMemcpy(nearest, d_nearest, n_density * sizeof (uint32_t), hipMemcpyDeviceToHost);
        tock_ms(2) times[2] = t_elapsed; tick(2)
        for(int64_t k=0; k<N; k++) {
            sites[k] = 0;
            count[k] = 0;
        }
        for(int64_t j=0; j<n_density; j++){ // Iterating on nearest
            sites[nearest[j]] += density[j];
            count[nearest[j]] ++;
        }
        #pragma omp parallel for shared(sites, count) schedule(static)
        for (int64_t k = 0; k < N; k++) {
            if (count[k] == 0) rand_complex(z1, z2, sites + k, 1);
            else sites[k] /= (double)count[k];
        }
        tock_ms(2) times[3] = t_elapsed;
        tock_ms(1)
        float m = 100.0f / t_elapsed;
        std::cout << "                   " << std::setw(2) << i+1
                  << " | " << std::setw(6) << std::setprecision(1) << t_elapsed
                  << " | " << std::setw(5) << std::setprecision(2) << times[0]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[1]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[2]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[3]*m << std::endl;
    }
    hipFree(d_density);
    hipFree(d_nearest);
    free(sites);
    free(density);
    free(nearest);
    free(count);
    tock_s(0)
    std::cout << "  :: total " << std::setprecision(3) << t_elapsed << 's' << std::endl;
    return d_sites;
}

__global__ void scale_complex(double real, double imag, complex_t offset, complex_t * data, uint64_t N) {
    auto increment = (uint64_t) blockDim.x * gridDim.x;
    for(uint64_t i=threadIdx.x+blockIdx.x*blockDim.x; i<N; i+=increment) {
        data[i].real(data[i].real() * real);
        data[i].imag(data[i].imag() * imag);
        data[i] += offset;
    }
}

/**
 * Updates the position of the sites. To be called after density points are sorted by nearest site
 * @param density_points
 * @param N_density
 * @param sites
 * @param N_sites
 * @param nearest
 */
__global__ void update_sites(
        complex_t * density_points, uint32_t N_density,
        complex_t * sites, uint32_t N_sites,
        uint32_t * nearest
) {
    auto site_index = threadIdx.x + blockIdx.x * blockDim.x;
    if(site_index >= N_sites) return;

    int64_t count = 0;
    complex_t sum = 0.0;
    uint32_t dpoint_index = lower_bound(site_index, nearest, N_density);
    while(nearest[dpoint_index] == site_index) {
        sum += density_points[dpoint_index];
        dpoint_index++;
        count++;
    }
    if(count > 0) sites[site_index] = sum / (double) count;
}

complex_t* particles_gpu(complex_t z1, complex_t z2, uint32_t N){
    int64_t n_density = 128*N;
    auto M = ((N-1) >> 10) + 1; // (n_density + 1023) / 1024 = (n_density-1)/ 2^(10)
    auto D = ((n_density-1) >> 10) + 1;

    complex_t *d_density, *d_sites;
    uint32_t *d_nearest;
    hipMalloc((void **)&d_density, n_density * sizeof (complex_t));
    hipMalloc((void **)&d_sites, N * sizeof (complex_t));
    hipMalloc((void **)&d_nearest, n_density * sizeof (uint32_t));

    PRINT_INITIAL timers(3) tick(0)
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);

    auto deltaReal = z2.real()-z1.real();
    auto deltaImag = z2.imag()-z1.imag();

    hiprandGenerateUniformDouble(gen, (double*) d_sites, N*2);
    scale_complex<<<128, 1024>>>(deltaReal, deltaImag, z1, d_sites, N);
    hiprandGenerateUniformDouble(gen, (double*) d_density, n_density*2);
    scale_complex<<<128, 1024>>>(deltaReal, deltaImag, z1, d_density, n_density);
    hipDeviceSynchronize();
    tock_ms(0) std::cout << " generated in " << t_elapsed << "ms" << std::endl;

    float times[3];
    std::cout << "Lloyd's algorithm:  i | t (ms) | n. c. | sortk | s. u." << std::endl << std::fixed;
    tick(0)
    for(int16_t i=0; i<20; i++){  // Iterating to convergence
        tick(1) tick(2)
        compute_nearest<<<D, 1024>>>(d_density, n_density, d_sites, N, d_nearest);
        hipDeviceSynchronize();
        tock_ms(2) times[0] = t_elapsed; tick(2)
        thrust::sort_by_key(thrust::device, d_nearest, d_nearest + n_density, d_density);
        tock_ms(2) times[1] = t_elapsed; tick(2)
        update_sites<<<M, 1024>>>(d_density, n_density, d_sites, N, d_nearest);
        hipDeviceSynchronize();
        tock_ms(2) times[2] = t_elapsed; tock_ms(1)
        float m = 100.0f / t_elapsed;
        std::cout << "                   " << std::setw(2) << i+1
                  << " | " << std::setw(6) << std::setprecision(1) << t_elapsed
                  << " | " << std::setw(5) << std::setprecision(2) << times[0]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[1]*m
                  << " | " << std::setw(5) << std::setprecision(2) << times[2]*m << std::endl;
    }
    tock_s(0) std::cout << "  :: total " << std::setprecision(3) << t_elapsed << 's' << std::endl;

    hipFree(d_density);
    hipFree(d_nearest);
    return d_sites;
}