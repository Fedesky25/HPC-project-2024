#include "hip/hip_runtime.h"
//
// Created by Sofyh02 on 10/08/2024.
//

#include "particle_generator.cuh"
#include <cstdlib>
#include <omp.h>
#include <cuda/std/cmath>
#include <random>
#include <chrono>


/**
 * Generates M random complex numbers in the rectangle
 * @param z1 lower-left vertex
 * @param z2 upper-right vertex
 * @param rdm array of points
 * @param M number of sites
 */
void rand_complex(complex_t z1, complex_t z2, complex_t * rdm, uint64_t M) {
    auto seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_real_distribution<double> distribution(0.0,1.0);
    for(uint64_t i=0; i<M; i++){
        rdm[i].real(real(z1) + distribution(generator)*(real(z2)-real(z1)));
        rdm[i].imag(imag(z1) + distribution(generator)*(imag(z2)-imag(z1)));
    }
}


complex_t* particles_serial(complex_t z1, complex_t z2, uint32_t N){

    auto sites = (complex_t*) malloc(N * sizeof(complex_t));

    auto n_density = 128*(int64_t)N;
    auto density = (complex_t*) malloc(n_density * sizeof(complex_t));
    rand_complex(z1, z2, density, n_density); // Random complex density points

    rand_complex(z1, z2, sites, N); // Random complex sites

    // Moving sites
    auto nearest = (uint32_t*) malloc(n_density * sizeof(uint32_t));// To save nearest sites
    for(uint16_t i=0; i<50; i++){  // Iterating to convergence
        for(uint64_t j=0; j<n_density; j++){ // Iterating on density points
            double current, min = INFINITY;
            for(uint64_t k=0; k<N; k++){  // Iterating on sites to save the nearest site to each density point
                current = cuda::std::norm(density[j]-sites[k]);
                if(current < min){
                   nearest[j] = k;
                   min = current;
               }
            }
        } // Here nearest[] has been filled in
        for(uint64_t k=0; k<N; k++){ // Iterating on sites
            double ctr = 0;
            sites[k] = 0;
            for(uint64_t j=0; j<n_density; j++){ // Iterating on nearest
                if(nearest[j] == k){  // Finding density points associated to the k-th site
                    sites[k] += density[j];
                    ctr++;
                }
            }
            if(ctr != 0) sites[k] /= ctr;
        }
    }
    free(density);
    free(nearest);
    return sites;
}


complex_t* particles_omp(complex_t z1, complex_t z2, uint32_t N){

    auto sites = (complex_t*) malloc(N * sizeof(complex_t));
    auto count = (int64_t*) malloc(N * sizeof(int64_t));
    PRINTLN("Generating " << N << " random sites");
    rand_complex(z1, z2, sites, N); // Random complex sites

    auto n_density = 128*(int64_t)N;
    auto density = (complex_t*) malloc(n_density * sizeof(complex_t));
    PRINTLN("Generating " << n_density << " density points")
    rand_complex(z1, z2, density, n_density); // Random complex density points
    auto nearest = (uint32_t*) malloc(n_density * sizeof(uint32_t));// To save nearest sites

    omp_set_num_threads(10);

    for(int16_t i=0; i<30; i++){  // Iterating to convergence
        PRINTLN("Iteration " << i+1)
        #pragma omp parallel for shared(nearest, density, sites) schedule(static)
        for (int64_t j = 0; j < n_density; j++) { // Iterating on density points
            double current, min = INFINITY;
            for (int64_t k = 0; k < N; k++) {  // Iterating on sites to save the nearest site to each density point
                current = cuda::std::norm(density[j] - sites[k]);
                if (current < min) {
                    nearest[j] = k;
                    min = current;
                }
            }
        }
        for(int64_t k=0; k<N; k++) {
            sites[k] = 0;
            count[k] = 0;
        }
        for(int64_t j=0; j<n_density; j++){ // Iterating on nearest
            sites[nearest[j]] += density[j];
            count[nearest[j]] ++;
        }

        #pragma omp parallel for shared(sites, count) schedule(static)
        for (int64_t k = 0; k < N; k++) {
            if (count[k] == 0) rand_complex(z1, z2, sites + k, 1);
            else sites[k] /= (double) count[k];
        }
    }
    free(density);
    free(nearest);
    free(count);
    return sites;
}

__global__ void compute_nearest(
        complex_t * density_points, int64_t N_density,
        complex_t * sites, uint32_t N_sites,
        uint32_t * nearest
){
    auto index = threadIdx.x + blockIdx.x*blockDim.x;
    if(index >= N_density) return;
    double current, min = INFINITY;
    complex_t z = density_points[index];
    uint32_t n;
    for (uint32_t k = 0; k < N_sites; k++) {  // Iterating on sites to save the nearest site to each density point
        current = cuda::std::norm(z - sites[k]);
        if (current < min) {
            n = k;
            min = current;
        }
    }
    nearest[index] = n;
}

complex_t* particles_mixed(complex_t z1, complex_t z2, uint32_t N){

    auto sites = (complex_t*) malloc(N * sizeof(complex_t));
    auto count = (int64_t*) malloc(N * sizeof(int64_t));
    PRINTLN("Generating " << N << " random sites");
    rand_complex(z1, z2, sites, N); // Random complex sites

    auto n_density = 128*(int64_t)N;
    auto density = (complex_t*) malloc(n_density * sizeof(complex_t));
    PRINTLN("Generating " << n_density << " density points")
    rand_complex(z1, z2, density, n_density); // Random complex density points

    auto nearest = (uint32_t*) malloc(n_density * sizeof(uint32_t));// To save nearest sites

    auto M = ((n_density-1) >> 10) + 1; // (n_density + 1023) / 1024 = (n_density-1)/ 2^(10)
    complex_t *d_density, *d_sites;
    int64_t *d_nearest;
    hipMalloc((void **)&d_density, n_density * sizeof (complex_t));
    hipMalloc((void **)&d_sites, N * sizeof (complex_t));
    hipMalloc((void **)&d_nearest, n_density * sizeof (int64_t));

    hipMemcpy(d_density, density, n_density * sizeof (complex_t), hipMemcpyHostToDevice);

    PRINT("Arranging particles: ");
    for(int16_t i=0; i<30; i++){  // Iterating to convergence
        hipMemcpy(d_sites, sites, N * sizeof (complex_t), hipMemcpyHostToDevice);
        compute_nearest<<<M, 1024>>>(d_density, n_density, d_sites, N, d_nearest);
        hipMemcpy(nearest, d_nearest, n_density * sizeof (int64_t), hipMemcpyDeviceToHost);

        for(int64_t k=0; k<N; k++) {
            sites[k] = 0;
            count[k] = 0;
        }
        for(int64_t j=0; j<n_density; j++){ // Iterating on nearest
            sites[nearest[j]] += density[j];
            count[nearest[j]] ++;
        }

        #pragma omp parallel for shared(sites, count) schedule(static)
        for (int64_t k = 0; k < N; k++) {
            if (count[k] == 0) rand_complex(z1, z2, sites + k, 1);
            else sites[k] /= (double)count[k];
        }
        PRINT(' ' << i+1)
    }
    PRINTLN(' ');
    hipFree(d_density);
    hipFree(d_nearest);
    free(density);
    free(nearest);
    free(count);
    return sites;
}