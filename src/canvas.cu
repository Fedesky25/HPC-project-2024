#include "hip/hip_runtime.h"
#include "canvas.cuh"

#define SATURATION 0.55
#define LIGHTNESS 0.55

#define BG_R 21
#define BG_G 21
#define BG_B 25

__device__ __host__ bool CanvasPixel::update_age(uint16_t _age) {
    bool ok = true;
    if(age == UINT16_MAX) age = _age;
    else if(age + multiplicity + 1 == _age) multiplicity++;
    else ok = false;
    return ok;
}

/**
 * @see https://stackoverflow.com/questions/2353211/hsl-to-rgb-color-conversion
 * @param t
 * @return
 */
__device__ __host__ double xToRGB(double t) {
    constexpr double q = LIGHTNESS + SATURATION - LIGHTNESS*SATURATION;
    constexpr double p = 2.0*LIGHTNESS - q;
    constexpr double f = (q-p)*2.0;

    if (t < 0.0) t += 3.0;
    if (t > 3.0) t -= 3.0;
    if (t < 0.5) return p + f * t;
    if (t < 1.5) return q;
    if (t < 2.0) return p + f * (2 - t);
    return p;
}

__device__ __host__ void CanvasPixel::set_color(double square_speed, double factor) {
    // saturation & lightness are fixed to 0.55
    auto x = 2.0 * square_speed / (square_speed + factor); // 1.0 / (1.0 + factor/ square_speed)
    // h = x / 3
    color.r = static_cast<uint8_t>(cuda::std::round(255.0 * xToRGB(x + 1)));
    color.g = static_cast<uint8_t>(cuda::std::round(255.0 * xToRGB(x)));
    color.b = static_cast<uint8_t>(cuda::std::round(255.0 * xToRGB(x - 1)));
}

__device__ __host__ ARGB CanvasPixel::get_color(uint16_t time) const {
    ARGB result;
    if(time < age || time >= age + multiplicity + 200) {
        result.r = BG_R;
        result.g = BG_G;
        result.b = BG_B;
    }
    else if(time <= age + multiplicity) result = color;
    else {
         auto x = (time - age - multiplicity) * 0.005;
         result.r = static_cast<uint8_t>(cuda::std::round(x*BG_R + (1-x)*color.r));
         result.g = static_cast<uint8_t>(cuda::std::round(x*BG_G + (1-x)*color.g));
         result.b = static_cast<uint8_t>(cuda::std::round(x*BG_B + (1-x)*color.b));
    }
    return result;
}

