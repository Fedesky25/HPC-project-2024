#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <cstdlib>
#include <iostream>


int32_t CanvasAdapter::where(complex_t z) const {
    auto row = static_cast<int32_t>(std::round(z.real() - center.real())) + (int32_t)(width >> 1);
    auto col = static_cast<int32_t>(std::round(z.imag() - center.imag())) + (int32_t)(height >> 1);
    if(row < 0 || row >= height || col < 0 || col > width) return -1;
    else return col + row * (int32_t)width;
}

std::ostream &operator<<(std::ostream &os, CanvasAdapter &cv) {
    return os << cv.width << 'x' << cv.height << '@' << cv.center << '$' << cv.scale << "px/u";
}

std::ostream &operator<<(std::ostream &os, EvolutionOptions &eo) {
    return os << eo.frame_count << "f, " << eo.ms_per_frame << "ms/f, dt=" << eo.delta_time << ", @v=" << eo.speed_factor;
}

void Configuration::bounds(complex_t *min, complex_t *max) const {
    auto extra = 2*margin*particle_distance;
    double dr = (canvas.width + extra) / canvas.scale * 0.5;
    double di = (canvas.height + extra) / canvas.scale * 0.5;
    min->real(canvas.center.real() - dr);
    min->imag(canvas.center.imag() - di);
    max->real(canvas.center.real() + dr);
    max->imag(canvas.center.imag() + di);
}

void Configuration::sizes(unsigned int *width, unsigned int *height) const {
    auto extra = 2*margin*particle_distance;
    *width = canvas.width + extra;
    *height = canvas.height + extra;
}

uint32_t Configuration::particle_number() const {
    auto extra = 2*margin*particle_distance;
    return (canvas.width + extra) * (canvas.height + extra) / (particle_distance*particle_distance);
}