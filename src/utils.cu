#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <cstdlib>
#include <iostream>


inline unsigned udist(unsigned a, unsigned b) {
    return (a > b) ? a-b : b-a;
}

void TilesCount::cover(unsigned int width, unsigned int height) {
    unsigned rev = 0;
    if(height > width) {
        rev = width;
        width = height;
        height = rev;
    }
    float min = INFINITY;
    float ratio = (float) width / (float) height;
    for(unsigned r=1; r <= 32; r++) {
        auto c = static_cast<unsigned>(std::round(ratio*r));
        while(r * c > 1024) c--;
        auto d = std::abs((float) c / (float) r - ratio);
        if(d <= min) {
            rows = r;
            cols = c;
            min = d;
        }
    }
    if(rev) {
        rev = rows;
        rows = cols;
        cols = rev;
    }
}

PixelIndex Canvas::where(complex_t z) const {
    auto row = static_cast<int32_t>(std::round(z.real() - center.real())) + (width >> 1);
    auto col = static_cast<int32_t>(std::round(z.imag() - center.imag())) + (height >> 1);
    if(row < 0 || row >= width || col < 0 || col > height) return {};
    else return { row, col };
}

std::ostream &operator<<(std::ostream &os, Canvas &cv) {
    return os << cv.width << 'x' << cv.height << '@' << cv.center << '$' << cv.scale << "px/u";
}

void Configuration::bounds(complex_t *min, complex_t *max) const {
    auto extra = 2*margin*particle_distance;
    double dr = (canvas.width + extra) / canvas.scale * 0.5;
    double di = (canvas.height + extra) / canvas.scale * 0.5;
    min->real(canvas.center.real() - dr);
    min->imag(canvas.center.imag() - di);
    max->real(canvas.center.real() + dr);
    max->imag(canvas.center.imag() + di);
}

unsigned long Configuration::particle_number() const {
    auto extra = 2*margin*particle_distance;
    return (canvas.width + extra) * (canvas.height + extra) / (particle_distance*particle_distance);
}

TilesCount::TilesCount(Configuration &config) {
    auto extra = config.margin*config.particle_distance*2;
    cover(config.canvas.width+extra, config.canvas.height+extra);
}
