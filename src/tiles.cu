#include "hip/hip_runtime.h"
#include "tiles.cuh"
#include "lower_bound.cuh"
#include "sorter.cuh"
#include <iomanip>


Tiles::Tiles(Configuration * config) {
    unsigned width, height;
    config->sizes(&width, &height);
    cover(width, height);
}

void Tiles::cover(unsigned int width, unsigned int height) {
    float r = sqrt((float) width / (float) height);
    cols = static_cast<uint_fast16_t>(32*r);
    rows = static_cast<uint_fast16_t>(32/r);
}

__global__ void compute_tile(
        uint32_t N, complex_t * particles, unsigned * tile_map,
        complex_t min, double hscale, double vscale,
        uint_fast16_t cols, uint_fast16_t rows
) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= N) return;
    auto c = static_cast<unsigned>(hscale * (particles[i].real() - min.real()));
    auto r = static_cast<unsigned>(vscale * (particles[i].imag() - min.imag()));
    tile_map[i] = ((c&1)+2*(r&1))*rows*cols + (r>>1)*cols + (c>>1);
}

__global__ void compute_offset_per_tile(uint32_t N, unsigned int * tile_map, uint32_t * offsets) {
    auto tile = threadIdx.x + blockIdx.x * blockDim.x;
    offsets[tile] = lower_bound(tile, tile_map, N);
}

uint32_t * Tiles::sort(complex_t &min, complex_t &max, complex_t *particles, uint32_t N) const {
    timers(2) tick(0)
    float times[4];
    uint32_t * offsets;
    auto hscale = 2 * cols / (max.real() - min.real());
    auto vscale = 2 * rows / (max.imag() - min.imag());
    auto tile_count = total();
    auto M = 1 + (N - 1)/tile_count;
    tick(1)
    KVSorter<unsigned, complex_t> tile_map(N, particles);
    hipMalloc(&offsets, (1+4*tile_count) * sizeof(uint32_t));
    hipDeviceSynchronize();
    tock_us(1) times[0] = t_elapsed; tick(1)
    compute_tile<<<M, tile_count>>>(N, particles, tile_map.keys(), min, hscale, vscale, cols, rows);
    hipDeviceSynchronize();
    tock_us(1) times[1] = t_elapsed; tick(1)
    // thrust::sort_by_key(thrust::device, tile_map, tile_map + N, particles);
    tile_map.sort();
    hipDeviceSynchronize();
    tock_us(1) times[2] = t_elapsed; tick(1)
    compute_offset_per_tile<<<4, tile_count>>>(N, tile_map.keys(), offsets);
    hipMemcpy(offsets + 4*tile_count, &N, sizeof(uint32_t), hipMemcpyHostToDevice);
    tock_us(1) times[3] = t_elapsed;
    tock_us(0)
    float m = 100.0f / t_elapsed;
    std::cout.precision(1);
    std::cout << "Particles sorted by tile in " << std::fixed << t_elapsed << "us (allocation: "
              << std::fixed << std::setprecision(1) << times[0]*m << "%, compute indexes: "
              << std::fixed << std::setprecision(1) << times[1]*m << "%, sort: "
              << std::fixed << std::setprecision(1) << times[2]*m << "%, compute offsets: "
              << std::fixed << std::setprecision(1) << times[3]*m << "%)" << std::endl;
    return offsets;
}
