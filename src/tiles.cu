#include "hip/hip_runtime.h"
#include "tiles.cuh"
#include "thrust/sort.h"
#include "lower_bound.cuh"

Tiles::~Tiles() {
    delete[] points;
    delete[] counts;
}

Tiles::Tiles(Configuration * config) {
    unsigned width, height;
    config->sizes(&width, &height);
    cover(width, height);
}

void Tiles::cover(unsigned int width, unsigned int height) {
    unsigned rev = 0;
    if(height > width) {
        rev = width;
        width = height;
        height = rev;
    }
    float min = INFINITY;
    float ratio = (float) width / (float) height;
    for(unsigned r=1; r <= 32; r++) {
        auto c = static_cast<unsigned>(std::round(ratio*r));
        while(r * c > 1024) c--;
        auto d = std::abs((float) c / (float) r - ratio);
        if(d <= min) {
            rows = r;
            cols = c;
            min = d;
        }
    }
    if(rev) {
        rev = rows;
        rows = cols;
        cols = rev;
    }
    delete[] counts;
    auto N = total();
    counts = new uint_fast16_t [N];
    for(uint_fast16_t i=0; i<N; i++) counts[i] = 0;
}

__global__ void compute_tile(
        uint32_t N, complex_t * particles, uint_fast16_t * tile_map,
        complex_t min, double hscale, double vscale, uint_fast16_t cols
) {
    auto i = threadIdx.x + blockIdx.x * blockDim.x;
    if(i >= N) return;
    auto c = static_cast<uint_fast16_t>(hscale * (particles[i].real() - min.real()));
    auto r = static_cast<uint_fast16_t>(vscale * (particles[i].imag() - min.imag()));
    tile_map[i] = c + r*cols;
}

__global__ void compute_offset_per_tile(uint32_t N, unsigned int * tile_map, uint32_t * offsets) {
    auto tile = threadIdx.x + blockIdx.x * blockDim.x;
    offsets[tile] = lower_bound(tile, tile_map, N);
}

uint32_t * Tiles::sort(complex_t &min, complex_t &max, complex_t *particles, uint32_t N) const {
    timers(2) tick(0)
    float times[4];
    unsigned int * tile_map;
    uint32_t * offsets;
    auto hscale = cols / (max.real() - min.real());
    auto vscale = rows / (max.imag() - min.imag());
    auto tile_count = total();
    auto M = 1 + (N - 1)/tile_count;
    tick(1)
    hipMalloc(&tile_map, N * sizeof(unsigned int));
    hipMalloc(&offsets, (1+tile_count) * sizeof(uint32_t));
    tock_us(1) times[0] = t_elapsed; tick(1)
    compute_tile<<<M, tile_count>>>(N, particles, tile_map, min, hscale, vscale, cols);
    tock_us(1) times[1] = t_elapsed; tick(1)
    thrust::sort_by_key(thrust::device, tile_map, tile_map + N, particles);
    tock_us(1) times[2] = t_elapsed; tick(1)
    auto block_dim = rows, grid_dim = cols;
    if(block_dim > grid_dim) {
        // for sure one between rows and cols is less than 32 i.e. the dimension of a warp
        block_dim = cols;
        grid_dim = rows;
    }
    compute_offset_per_tile<<<grid_dim, block_dim>>>(N, tile_map, offsets);
    hipMemcpy(offsets + tile_count, &N, sizeof(uint32_t), hipMemcpyHostToDevice);
    tock_us(1) times[3] = t_elapsed;
    hipFree(tile_map);
    tock_us(0)
    float m = 100.0f / t_elapsed;
    std::cout.precision(1);
    std::cout << "Particles sorted by tile in " << std::fixed << t_elapsed << "us {alloc: " << times[0]*m
              << ", comp: " << times[1]*m << ", sort: " << times[2]*m << ", offsets: " << times[3]*m << '}' << std::endl;
    return offsets;
}
