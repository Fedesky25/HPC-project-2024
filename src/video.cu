//
// Created by feder on 27/09/2024.
//

#include "video.cuh"
#include "frames.cuh"
#include <fstream>
#include <iomanip>
#include <omp.h>

#define HEADER std::cout << "Frame computation: iter. | c (us) | w (ms)" << std::endl;

template<bool opaque>
void write_video_serial_internal(
        const char * filename, Canvas canvas,
        uint32_t frame_size, int32_t frame_count,
        const RGBA & background
) {
    constexpr auto bytes = opaque ? 3 : 4;
    auto mem = bytes * frame_size;
    auto frame = new unsigned char [mem];
    std::ofstream out(filename);
    RGBA color;
    unsigned char bg_bytes[bytes];
    background.write<opaque>(bg_bytes);
    auto inv_lifetime = 1.0f / (float) frame_count;
    float tc, tw;
    HEADER
    timers(2)
    tick(0)
    for(int32_t t=0; t<frame_count; t++) {
        tick(1)
        for(uint32_t i=0; i<frame_size; i++) {
            auto delta = canvas[i].time_distance(t, frame_count);
            if(delta >= frame_count + canvas[i].multiplicity) {
                for(int b=0; b<bytes; b++) frame[bytes*i + b] = bg_bytes[b];
            }
            else {
                color.from_hue(canvas[i].hue);
                if(delta < canvas[i].multiplicity) color.A = 1.0f;
                else {
                    color.A = (float) (frame_count+canvas[i].multiplicity-delta) * inv_lifetime;
                    color.over<opaque>(&background);
                }
                color.write<opaque>(frame + bytes*i);
            }
        }
        tock_ms(1)
        tc = t_elapsed;
        tick(1)
        out.write(reinterpret_cast<const char *>(frame), mem);
        tock_ms(1)
        tw = t_elapsed;
        std::cout << "                   " << std::setw(5) << (t+1)
                  << " | " << std::setw(6) << std::setprecision(2) << tc
                  << " | " << std::setw(6) << std::setprecision(2) << tw << std::endl;
    }
    tock_s(0)
    std::cout << "  :: total " << t_elapsed << 's' << std::endl;
    delete [] frame;
}

void write_video_serial(
        const char * filename, Canvas canvas,
        uint32_t frame_size, int32_t frame_count,
        const RGBA & background
) {
    if(background.A == 1.0f) write_video_serial_internal<true>(filename, canvas, frame_size, frame_count, background);
    else write_video_serial_internal<false>(filename, canvas, frame_size, frame_count, background);
}


template<bool opaque>
void write_video_omp_internal(
        const char * filename,
        const Canvas * canvases, uint32_t canvas_count,
        uint32_t frame_size, int32_t frame_count,
        const RGBA & background
) {
    std::ofstream out(filename);
    constexpr auto bytes = opaque ? 3 : 4;
    auto mem = bytes * frame_size;
    unsigned char * frame_buffers[2];
    frame_buffers[0] = new unsigned char [mem];
    frame_buffers[1] = new unsigned char [mem];
    unsigned char bg_bytes[bytes];
    background.write<opaque>(bg_bytes);
    auto inv_lifetime = 1.0f / (float) frame_count;
    float tc, tw=NAN;

    int32_t frame_size_signed = frame_size;

    omp_set_nested(1);

    std::cout << "Frame computation: iter. | c (ms) | w (ms)" << std::endl;
    auto start_all = std::chrono::steady_clock::now();
    for(int32_t t=0; t<frame_count; t++) {
        #pragma omp parallel sections
        {
            #pragma omp section
            {
                if(t > 0) {
                    auto start = std::chrono::steady_clock::now();
                    out.write(reinterpret_cast<const char *>(frame_buffers[(t-1)&1]), mem);
                    auto end = std::chrono::steady_clock::now();
                    tw = (std::chrono::duration<float, std::milli>(end-start)).count();
                }
            }
            #pragma omp section
            {
                auto start = std::chrono::steady_clock::now();
                auto frame = frame_buffers[t&1];
                #pragma omp parallel
                {
                    RGBA color;
                    CanvasPixel * pixel;
                    int32_t delta_min, delta;
                    #pragma omp for schedule(static)
                    for(int32_t i=0; i<frame_size_signed; i++) {
                        pixel = &canvases[0][i];
                        delta_min = canvases[0][i].time_distance(t, frame_count);
                        for(uint32_t c=1; c<canvas_count; c++) {
                            delta = canvases[c][i].time_distance(t, frame_count);
                            if(delta < delta_min) {
                                delta_min = delta;
                                pixel = &canvases[c][i];
                            }
                        }
                        if(delta >= frame_count + pixel->multiplicity) {
                            for(int b=0; b<bytes; b++) frame[bytes*i + b] = bg_bytes[b];
                        }
                        else {
                            color.from_hue(pixel->hue);
                            if(delta < pixel->multiplicity) color.A = 1.0f;
                            else {
                                color.A = (float) (frame_count+pixel->multiplicity-delta) * inv_lifetime;
                                color.over<opaque>(&background);
                            }
                            color.write<opaque>(frame + bytes*i);
                        }
                    }
                }
                auto end = std::chrono::steady_clock::now();
                tc = (std::chrono::duration<float, std::milli>(end-start)).count();
            }
        }
        std::cout << "                   " << std::setw(5) << (t+1)
                  << " | " << std::setw(6) << std::setprecision(2) << tc
                  << " | " << std::setw(6) << std::setprecision(2) << tw << std::endl;
    }
    auto end_all = std::chrono::steady_clock::now();
    float total = (std::chrono::duration<float, std::ratio<1>>(end_all-start_all)).count();
    std::cout << "  :: total " << total << 's' << std::endl;

    delete [] frame_buffers[0];
    delete [] frame_buffers[1];
}


void write_video_omp(
        const char * filename,
        const Canvas * canvases, uint32_t canvas_count,
        uint32_t frame_size, int32_t frame_count,
        const RGBA & background
) {
    if(background.A == 1.0f) write_video_omp_internal<true>(filename, canvases, canvas_count, frame_size, frame_count, background);
    else write_video_omp_internal<false>(filename, canvases, canvas_count, frame_size, frame_count, background);
}



void write_video_gpu(
        const char * filename,
        const Canvas * canvases, uint32_t canvas_count,
        uint32_t frame_size, int32_t frame_count,
        const FixedHSLA * background
) {
    std::ofstream raw_output(filename);
    auto frame_mem = frame_size * sizeof(uint32_t);
    uint32_t *h_frame, *d_frame[2];
    h_frame = (uint32_t*) malloc(frame_mem);
    hipMalloc(d_frame, frame_mem);
    hipMalloc(d_frame+1, frame_mem);
    std::cout << "Frame buffers: CPU=" << (((frame_mem-1)>>20)+1) << "MB, GPU="
              << (((frame_mem*2-1)>>20)+1) << "MB" << std::endl << std::fixed;
    std::cout << "Frame computation: iter. | c (us) | w (ms)" << std::endl;
    std::cout.width(6);

    float time_write, time_compute;
    auto begin = std::chrono::steady_clock::now();
    compute_frame_gpu(
            0, frame_count,
            canvases, canvas_count,
            d_frame[0], frame_size,
            background);
    hipDeviceSynchronize();
    auto _end = std::chrono::steady_clock::now();
    time_compute = (std::chrono::duration<float,std::micro>(_end-begin)).count();
    std::cout << "                   " << std::setw(5) << 0
              << " | " << std::setw(6) << time_compute
              << " | " << std::endl;


    for(int32_t i=1; i<frame_count; i++) {
        #pragma omp parallel sections num_threads(2)
        {
            #pragma omp section
            {
                auto start = std::chrono::steady_clock::now();
                hipMemcpy(h_frame, d_frame[(i&1)^1], frame_mem, hipMemcpyDeviceToHost);
                raw_output.write(reinterpret_cast<const char *>(h_frame), frame_mem);
                auto end = std::chrono::steady_clock::now();
                time_write = (std::chrono::duration<float, std::milli>(end - start)).count();
            }
            #pragma omp section
            {
                auto start = std::chrono::steady_clock::now();
                compute_frame_gpu(i, frame_count, canvases, canvas_count, d_frame[i&1], frame_size, background);
                hipDeviceSynchronize();
                auto end = std::chrono::steady_clock::now();
                time_compute = (std::chrono::duration<float,std::micro>(end-start)).count();
            }
        }
        std::cout << "                   " << std::setw(5) << i
                  << " | " << std::setw(6) << time_compute
                  << " | " << std::setw(6) << time_write << std::endl;
    }

    begin = std::chrono::steady_clock::now();
    hipMemcpy(h_frame, d_frame[(frame_count-1)&1], frame_mem, hipMemcpyDeviceToHost);
    raw_output.write(reinterpret_cast<const char *>(h_frame), frame_mem);
    _end = std::chrono::steady_clock::now();
    time_write = (std::chrono::duration<float,std::milli>(_end-begin)).count();
    std::cout << "                   " << std::setw(5) << frame_count
              << " |        | " << std::setw(6) << time_write << std::endl;

    hipFree(d_frame[0]);
    hipFree(d_frame[1]);
    free(h_frame);
}