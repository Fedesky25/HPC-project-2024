#include "hip/hip_runtime.h"
//
// Created by Sofyh02 on 15/08/2024.
//
#include "evolution.cuh"
#include "lower_bound.cuh"

__global__ void evolve(Canvas* canvas, CanvasAdapter * adapter, EvolutionOptions options, complex_t* particles,
                       uint32_t* belonging_tile, uint32_t* count,
                       complex_t (*func)(complex_t, FnVariables*), FnVariables* variables
                       ){
    auto tile_idx = threadIdx.x + threadIdx.y * blockDim.x;
    auto canvas_idx = blockIdx.x + blockIdx.y * gridDim.x;

    if(canvas_idx >= count[tile_idx]) return;

    auto particle_idx = lower_bound(tile_idx, belonging_tile, blockDim.x * blockDim.y);
    particle_idx += canvas_idx;
    auto z = particles[particle_idx];

    complex_t v, dz;
    double D, elapsed;
    auto dt = options.delta_time;
    auto steps = options.frame_count;
    // Evolving particle looping on lifetime (steps)
    for(uint32_t j=0; j<steps; j++){
        elapsed = 0.0;
        do {
            v = func(z, variables);
            dz = v * dt;
            D = (adapter->scale * cuda::std::abs(dz));
            if(D > 1) {
                dz /= D;
                elapsed += dt/D;
            }
            else{
                elapsed += dt;
            }
            auto pixel_idx = adapter->where(z);
            if(pixel_idx != -1) {
                canvas[canvas_idx][pixel_idx].update_age(j);
                canvas[canvas_idx][pixel_idx].set_color(cuda::std::norm(v), options.speed_factor);
            }
            z += dz;
        } while(elapsed < dt);
    }
}
