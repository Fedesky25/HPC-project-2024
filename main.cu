#include "hip/hip_runtime.h"
#include <iostream>
#include "utils.cuh"
#include "cli.hpp"
#include "getopt.h"
#include "tiles.cuh"
#include "particle_generator.cuh"
#include "fstream"
#include "complex_functions.cuh"
#include "canvas.cuh"
#include "omp.h"


int main(int argc, char * argv[]) {
    if(argc < 2) {
        print_usage();
        return 1;
    }

    Configuration config;
    auto err = parse_args(argc, argv, &config);
    if(err) return 1;

    if(optind >= argc) {
        std::cerr << "Missing function to plot" << std::endl;
        return 1;
    }
    auto fn_choice = strtofn(argv[optind]);
    if(fn_choice == FunctionChoice::NONE) {
        std::cerr << "Function string name not recognized" << std::endl;
        return 1;
    }

    std::cout << "Configuration:" << std::endl;
    std::cout << "  Output file: " << config.output << std::endl;
    std::cout << "  Complex numbers: " << config.vars.z[0] << ' ' << config.vars.z[1] << ' ' << config.vars.z[2] << std::endl;
    std::cout << "  Real and int numbers: " << config.vars.x << ", " << config.vars.n << std::endl;
    std::cout << "  Canvas: " << config.canvas << std::endl;
    std::cout << "  Evolution: " << config.evolution << std::endl;

    complex_t min, max;
    config.bounds(&min, &max);
    uint64_t N = config.particle_number();

    complex_t * points;
    uint32_t canvas_count;

    switch (config.mode) {
        case ExecutionMode::Serial:
            points = particles_serial(min, max, N);
            break;
        case ExecutionMode::OpenMP:
        {
            points = particles_omp(min, max, N);
            canvas_count = omp_get_max_threads();
            auto canvases = create_canvas_host(canvas_count, &config.canvas);
            break;
        }
        case ExecutionMode::GPU:
        {
            Tiles tiles(&config);
            unsigned tiles_count = tiles.total();
            std::cout << "  Tiles: " << tiles.rows << 'x' << tiles.cols << '=' << tiles_count << " with "
                      << (float) N / (float) tiles_count << " particles each" << std::endl;
            points = particles_gpu(min, max, N);
            uint_fast16_t * tile_map;
            uint32_t * count_per_tile;
            tiles.sort(min, max, points, N, &tile_map, &count_per_tile);
            canvas_count = get_canvas_count_serial(count_per_tile, tiles_count);
            auto canvases = create_canvas_device(canvas_count, &config.canvas);
            break;
        }
    }


    return 0;
}
